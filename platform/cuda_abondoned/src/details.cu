#include <iostream>
#include <stdio.h>
#include "renderInfo.hpp"
#include "sobol.hpp"
#include <hip/hip_runtime.h>
namespace Renderer
{
    namespace Cuda
    {   
        Vec3*   initVertex() {
            Vec3* tmp;
            hipMalloc((void**)&tmp, vertexBuffer.size()*sizeof(Vec3));
            hipMemcpy(tmp, &vertexBuffer[0],
            vertexBuffer.size()*sizeof(Vec3), hipMemcpyHostToDevice);
            return tmp;
        }

        TextureInfo* initTexture() {
            TextureInfo* tmp;
            hipMalloc((void**)&tmp,
            textureBuffer.size()*sizeof(TextureInfo));
            hipMemcpy(tmp, &textureBuffer[0],
            textureBuffer.size()*sizeof(TextureInfo), hipMemcpyHostToDevice);
            return tmp;
        }

        MaterialInfo* initMaterial() {
            MaterialInfo* tmp;
            hipMalloc((void**)&tmp,
            materialBuffer.size()*sizeof(MaterialInfo));
            hipMemcpy(tmp, &materialBuffer[0],
            materialBuffer.size()*sizeof(MaterialInfo), hipMemcpyHostToDevice);
            return tmp;
            //hipMemcpyToSymbol(HIP_SYMBOL(gpuMaterialBuffer), &tmp, sizeof(MaterialInfo*));
        }

        ObjectInfo* initObject () {
            ObjectInfo* tmp;
            hipMalloc((void**)&tmp,
            objectBuffer.size()*sizeof(ObjectInfo));
            hipMemcpy(tmp, &objectBuffer[0],
            objectBuffer.size()*sizeof(ObjectInfo), hipMemcpyHostToDevice);
            //hipMemcpyToSymbol(HIP_SYMBOL(gpuObjectBuffer), &tmp, sizeof(ObjectInfo*));
            return tmp;
        }
    }; // namespace Cuda
    
}; // namespace Renderer

#include "tasks.hpp"
#include "sobol.hpp"
#include "cudaRenderer.hpp"
#include "details.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

namespace Renderer
{

namespace Cuda
{

    inline hipError_t checkCudaError() {
        auto error = hipGetLastError();
        if (error != 0)
            cout<<"hipError_t:"<<error<<" - "<<hipGetErrorString(error)<<endl;
        return error;
    }

    bool checkCudaSupport() {
        int nDevices;
        hipGetDeviceCount(&nDevices);
        if (nDevices == 0) return false;
        return true;
    }

    inline const RenderConfig& getRenderConfig() {
        return renderConfig;
    }

    ErrorCode cudaRender(Vec3** pixels) {
        if (globalEnv == RenderEnv::UNDEFINE) {
            return ErrorCode::NOT_INIT;
        }
        else if (globalEnv != RenderEnv::CUDA) {
            return ErrorCode::ENV_NOT_SUPPORT;
        }

        auto width = getRenderConfig().width;
        auto height = getRenderConfig().height;

        // check device
        hipDeviceProp_t device;
        hipGetDeviceProperties(&device, 0);
        auto maxThreadPerBlock = device.maxThreadsPerBlock;
        // auto sharedMenPerBlockKb = device.sharedMemPerBlock;
        
        Renderer::Cuda::initSobolSequence();
        initRayGenerator();
        //hipMemcpyToSymbol(HIP_SYMBOL(gpuSobolSequence), sobolSequence,
        //    sizeof(Vec3)*SOBOL_SEQUENCE_CYCLE);
        // TODO: Compute Bounding box

        Vec3* generatedPixels;
        hipMallocManaged(&generatedPixels, sizeof(Vec3)*width*height);

        dim3 blockNum(width, height);
        dim3 threadNum(getRenderConfig().sampleNums);

        renderTask<<<blockNum, threadNum>>>(generatedPixels, width, height, getRenderConfig().depth,initTexture(), initMaterial(), initObject(), initVertex(), objectBuffer.size());

        hipDeviceSynchronize();

        if(checkCudaError()!=0) return ErrorCode::CUDA_ERROR;

        if(renderConfig.gamma) {
            // Wrap optimize
            gammaTask<<<blockNum, 1>>>(generatedPixels, width, height);
            hipDeviceSynchronize();
        }

        // render result
        *pixels = generatedPixels;

        cout<<"Render Finished in cuda"<<endl;
        return ErrorCode::SUCCESS;
    }

};
};
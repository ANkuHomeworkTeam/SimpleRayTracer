#include "hip/hip_runtime.h"
#include "sobol.hpp"
#include "maths/random.hpp"
#include "maths/operation.hpp"
#include <hiprand/hiprand_kernel.h>
#include <time.h>

namespace Renderer
{
    namespace Cuda
    {
        __constant__
        Vec3 gpuSobolSequence[SOBOL_SEQUENCE_CYCLE];
        __constant__
        Vec3 gpuSobolSequenceNormalized[SOBOL_SEQUENCE_CYCLE];

        __device__
        hiprandState *cState;

        __global__
        void initState(unsigned long seed, hiprandState* s) {
            int id = threadIdx.x;
            hiprand_init(seed, id, 0, &s[id]);
        }

        void initRandom(int threadNum) {
#       pragma region INIT_SOBOL
            hipMemcpyToSymbol(HIP_SYMBOL(gpuSobolSequence), sobolSequence,
            sizeof(Vec3)*SOBOL_SEQUENCE_CYCLE);
            Vec3* normalized = new Vec3[SOBOL_SEQUENCE_CYCLE];
            for(int i=0; i<SOBOL_SEQUENCE_CYCLE; i++) {
                auto v = sobolSequence[i] - 0.5;
                if (length(v) < 0.001f) v = { 1, 1, 1 };
                normalized[i] = normalize(v);
            }
            hipMemcpyToSymbol(HIP_SYMBOL(gpuSobolSequenceNormalized), normalized,
            sizeof(Vec3)*SOBOL_SEQUENCE_CYCLE);
#       pragma endregion INIT_SOBOL

#       pragma region INIT_RANDOM
            hiprandState* tmp;
            hipMalloc((void**)&tmp, sizeof(hiprandState)*threadNum);
            initState<<<1, threadNum>>>(time(NULL), tmp);
            hipMemcpyToSymbol(HIP_SYMBOL(cState), &tmp, sizeof(hiprandState*));
#       pragma endregion
        }

        __device__
        Vec3 getSobol(int index) {
            return gpuSobolSequence[index%SOBOL_SEQUENCE_CYCLE];
        }
        __device__
        Vec3 getSobolNormalized(int index) {
            __shared__
            static int counter;
            atomicAdd(&counter, 1);
            return gpuSobolSequenceNormalized[(index+counter)%SOBOL_SEQUENCE_CYCLE];
        }
        __device__
        float getRandom() {
            return hiprand_uniform(&cState[threadIdx.x]);
        }

        __device__
        Vec3 getRandomNormalizedVec3() {
            return normalize({getRandom() - 0.5f, getRandom() - 0.5f, getRandom() - 0.5f});
        }
    } // namespace Cuda
} // namespace Renderer

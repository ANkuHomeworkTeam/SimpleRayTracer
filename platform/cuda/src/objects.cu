#include "objects.hpp"
#include <iostream>
namespace Renderer
{
    namespace Cuda
    {
        __device__
        ObjectInfo* gpuObjectBuffer;
        __device__
        Vec3* gpuVertexBuffer;
        __device__
        int objectNum;

        void initObjects() {
            Vec3* tmpV;
            hipMalloc((void**)&tmpV, sizeof(Vec3)*vertexBuffer.size());
            hipMemcpy(tmpV, &vertexBuffer[0], sizeof(Vec3)*vertexBuffer.size(), hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(gpuVertexBuffer), &tmpV, sizeof(Vec3*));
            ObjectInfo* tmpO;
            hipMalloc((void**)&tmpO, sizeof(ObjectInfo)*objectBuffer.size());
            hipMemcpy(tmpO, &objectBuffer[0], sizeof(ObjectInfo)*objectBuffer.size(), hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(gpuObjectBuffer), &tmpO, sizeof(ObjectInfo*));
            int size = objectBuffer.size();
            hipMemcpyToSymbol(HIP_SYMBOL(objectNum), &size, sizeof(int));
            // hipMemcpy
        }

        __device__
        ObjectInfo getObject(unsigned int index) {
            return gpuObjectBuffer[index];
        }
        __device__
        Vec3 getVertex(unsigned int index) {
            return gpuVertexBuffer[index];
        }

        __device__
        int getObjectNum() {
            return objectNum;
        }
    } // namespace Cuda
} // namespace Renderer

#include "cudaRenderer.hpp"
#include "tasks.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;
using namespace Renderer;

bool Cuda::checkCudaSupport() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    if (nDevices == 0) return false;
    return true;
}

ErrorCode Cuda::cudaRender(Vec3** pixels) {
    if (globalEnv == RenderEnv::UNDEFINE) {
        return ErrorCode::NOT_INIT;
    }
    else if (globalEnv != RenderEnv::CUDA) {
        return ErrorCode::ENV_NOT_SUPPORT;
    }

    int height = renderConfig.height;
    int width  = renderConfig.width;

    // check device
    hipDeviceProp_t device;
    hipGetDeviceProperties(&device, 0);
    auto maxThreadPerBlock = device.maxThreadsPerBlock;
    auto sharedMenPerBlockKb = device.sharedMemPerBlock;

    // check end

    TextureInfo*    gpuTextureBuffer;
    MaterialInfo*   gpuMaterialBuffer;
    ObjectInfo*     gpuObjectBuffer;

    hipMalloc((void**)&gpuTextureBuffer,
        textureBuffer.size()*sizeof(TextureInfo));
    hipMalloc((void**)&gpuMaterialBuffer,
        materialBuffer.size()*sizeof(MaterialInfo));
    hipMalloc((void**)&gpuObjectBuffer,
        objectBuffer.size()*sizeof(ObjectInfo));

    hipMemcpy(gpuTextureBuffer, &textureBuffer[0],
        textureBuffer.size()*sizeof(TextureInfo), hipMemcpyHostToDevice);
    hipMemcpy(gpuMaterialBuffer, &materialBuffer[0],
        materialBuffer.size()*sizeof(MaterialInfo), hipMemcpyHostToDevice);
    hipMemcpy(gpuObjectBuffer, &objectBuffer[0],
        objectBuffer.size()*sizeof(ObjectInfo), hipMemcpyHostToDevice);

    Vec3* generatedPixels;
    hipMallocManaged(&generatedPixels, sizeof(Vec3)*width*height);

    dim3 blockNum(height, width);
    dim3 threadNum(256);
    renderTask<<<blockNum, threadNum>>>(generatedPixels, width, height);

    hipDeviceSynchronize();

    *pixels = generatedPixels;


    return ErrorCode::SUCCESS;
}


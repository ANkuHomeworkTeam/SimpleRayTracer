#include "BSDFs.hpp"

namespace Renderer
{
    namespace Cuda
    {
        __device__ MaterialInfo* gpuMaterialBuffer;
        __device__ TextureInfo* gpuTextureBuffer;

        void initMaterial() {
            MaterialInfo* tmp;
            hipMalloc((void**)&tmp, sizeof(MaterialInfo)*materialBuffer.size());
            hipMemcpy(tmp, &materialBuffer[0], sizeof(MaterialInfo)*materialBuffer.size(), hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(gpuMaterialBuffer), &tmp, sizeof(MaterialInfo*));    
        }
        void initTexture() {
            TextureInfo* tmp;
            hipMalloc((void**)&tmp, sizeof(TextureInfo)*textureBuffer.size());
            hipMemcpy(tmp, &textureBuffer[0], sizeof(TextureInfo)* textureBuffer.size(), hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(gpuTextureBuffer), &tmp, sizeof(TextureInfo*));
        }

        __device__
        MaterialInfo getMaterial(unsigned int index) {
            return gpuMaterialBuffer[index];
        }
        __device__
        TextureInfo getTexture(unsigned int index) {
            return gpuTextureBuffer[index];
        }

    } // namespace Cuda
} // namespace Renderer

#include "hip/hip_runtime.h"
#include "BSDFs.hpp"
#include "maths/random.hpp"
#include <stdio.h>
#include "objects.hpp"

namespace Renderer
{
    namespace Cuda
    {
        __device__ MaterialInfo* gpuMaterialBuffer;
        __device__ TextureInfo* gpuTextureBuffer;

        void initMaterial() {
            MaterialInfo* tmp;
            hipMalloc((void**)&tmp, sizeof(MaterialInfo)*materialBuffer.size());
            hipMemcpy(tmp, &materialBuffer[0], sizeof(MaterialInfo)*materialBuffer.size(), hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(gpuMaterialBuffer), &tmp, sizeof(MaterialInfo*));    
        }
        void initTexture() {
            TextureInfo* tmp;
            hipMalloc((void**)&tmp, sizeof(TextureInfo)*textureBuffer.size());
            hipMemcpy(tmp, &textureBuffer[0], sizeof(TextureInfo)* textureBuffer.size(), hipMemcpyHostToDevice);
            hipMemcpyToSymbol(HIP_SYMBOL(gpuTextureBuffer), &tmp, sizeof(TextureInfo*));
        }

        __device__
        const MaterialInfo& getMaterial(unsigned int index) {
            return gpuMaterialBuffer[index];
        }
        __device__
        const TextureInfo& getTexture(unsigned int index) {
            return gpuTextureBuffer[index];
        }

#   pragma region SHADE_AND_BSDF
        __device__
        Vec3 sampleTexture(id_t texture, float u, float v) {
            const TextureInfo& tx = getTexture(texture);
            switch (tx.type)
            {
            case TextureType::Solid:
                return tx.v1;
            default:
                return {1, 1, 1};
            }
        }


        __device__ inline
        Vec3 emitted(id_t material, float t, float maxDistance) {
            const MaterialInfo& m = getMaterial(material);
            float d = maxDistance - t;
            d = d > 0.f ? d : 0.f;
            return sampleTexture(m.texture, 0, 0) * m.luminance * std::pow(d, m.luminanceAttenuation);
        }

        __device__
        Vec3 BRDF(id_t material, MaterialType type, const Vec3& hitPoint, const Vec3& in, const Vec3& out) {
            const MaterialInfo& m = getMaterial(material);
            switch (type)
            {
            case MaterialType::LAMBERTAIN:
            case MaterialType::SPECULAR:
                return sampleTexture(m.texture, 0, 0) / M_PI;
            default:
                break;
            }
            
            return { 0, 0, 0 };
        }

        __device__
        Scattered shade(id_t material, const Ray& ray, float t, const Vec3& hitPoint, const Vec3& normal) {
            const MaterialInfo& m = getMaterial(material);
            Vec3 out = { 0, 0, 0 };
            Vec3 in = -ray.direction;
            Vec3 sample = {0, 0, 0};
            bool isDirect = false;
            id_t directObj = 0;
            switch (m.type)
            {
            case MaterialType::LAMBERTAIN:
                if (getRandom() > 0.45f) {
                    out = normalize(normal + getRandomNormalizedVec3());
                    sample = BRDF(material, MaterialType::LAMBERTAIN, hitPoint, in, out) * M_PI *  2 * cos(out, normal);
                }
                else {
                    isDirect = true;
                    auto lightSampling = sampleRandomLight(hitPoint, normal);
                    if (lightSampling) {   
                        out = normalize(lightSampling->samplePoint - hitPoint);
                        directObj = lightSampling->objId;
                        sample = BRDF(material, MaterialType::LAMBERTAIN, hitPoint, in, out) * M_PI;
                    }
                }
                break;
            case MaterialType::PHONG:
                if (getRandom() > 0.5) {

                }
                else {
                    
                }
                break;
            case MaterialType::EMITTED:
                break;
            case MaterialType::SPECULAR:
                out = normalize((ray.direction - 2*dot(ray.direction, normal)*normal))+getSobolNormalized(threadIdx.x)*m.glossy;
                sample = BRDF(material, MaterialType::SPECULAR, hitPoint, in, out) * M_PI ;// (cos(out, normal)*0.9 + 0.1);
                break;
            default:
                break;
            }
            Vec3 attenuation = emitted(material, t, m.luminanceDistance) + sample;
            // printf("sample: [ %f, %f, %f ]\n", attenuation.x, attenuation.y, attenuation.z);
            return {
                attenuation,
                Ray{hitPoint, out},
                isDirect,
                directObj
            };
        }

#   pragma endregion
    } // namespace Cuda
} // namespace Renderer
